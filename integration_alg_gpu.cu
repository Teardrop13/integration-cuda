#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>


#include <math.h>

long maxGridSize;
long maxThreadsPerBlock;

float *d_x_list;
float *d_y_list;
long long *d_length;
float *d_result;
float *d_result_list;

__global__ void integrate(float *d_x_list, float *d_y_list, float *d_result_list, long long *d_length) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < *(d_length)-1) {
        d_result_list[i] = (d_y_list[i] + d_y_list[i+1]) * (d_x_list[i+1] - d_x_list[i]) / 2;
    }
}

__global__ void sum_array(float *d_list, long long *d_length, float *d_result) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < (*d_length)-1) {
        atomicAdd(d_result, d_list[i]);
    }
}

void cuda_initialize() {
    int device = 0;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    maxGridSize = deviceProp.maxGridSize[0];
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    std::cout << "==============================================" << std::endl;
    std::cout << "Max dimension size of a grid size (x): " << maxGridSize << std::endl;
    std::cout << "Maximum number of threads per block: " << maxThreadsPerBlock << std::endl;
    std::cout << "==============================================" << std::endl << std::endl;
}

void cuda_clean() {
    hipFree(d_x_list);
    hipFree(d_y_list);
    hipFree(d_result_list);
    hipFree(d_length);
    hipFree(d_result);
}

float gpu_integrate(float *x_list, float *y_list, long long length) {
    cuda_initialize();

    float result = 0;

    hipMalloc((void **)&d_x_list, sizeof(long long) * length);
    hipMemcpy(d_x_list, x_list, sizeof(long long) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_y_list, sizeof(float) * length);
    hipMemcpy(d_y_list, y_list, sizeof(float) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_result_list, sizeof(float) * (length - 1));

    hipMalloc((void **)&d_length, sizeof(long long));
    hipMemcpy(d_length, &length, sizeof(long long), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_result, sizeof(float));
    hipMemcpy(d_result, &result, sizeof(float), hipMemcpyHostToDevice);

    long long threads = 256;
    long long blocks = ceil(1.* length / threads);

    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    integrate<<<blocks, threads>>>(d_x_list, d_y_list, d_result_list, d_length);

    hipDeviceSynchronize();

    sum_array<<<blocks, threads>>>(d_result_list, d_length, d_result);

    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << "gpu integration time: " << time << "ms" << std::endl;

    hipMemcpy(&result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    cuda_clean();

    return result;
}